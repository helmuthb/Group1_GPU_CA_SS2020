#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "matrix_graph.hpp"
#include "generator.hpp"
#include "graph.hpp"
#include "kernel.cu"
#include "array_structure_helper.cu"

#include <chrono>

#define NUM_RUNS 1

using namespace std;

void print_result(uint32_t * outbound, uint32_t *inbound, uint32_t *weights, uint32_t V) {
	cout << "H " << V << " " << V - 1 << " " << 1 << endl;
	for (int i = 0; i < V; i++) {
		if (weights[i] != UINT32_MAX) {
			cout << "E " << outbound[i] << " " << inbound[i] << " " << weights[i] << endl;
		}
	}
}

int main()
{
	MatrixGraph aGraph;

	chrono::steady_clock::time_point begin, end;
	double runtime;

	for (int i = 0; i < NUM_RUNS; i++) {

		uint2 * inbound_vertices, *outbound_vertices, *shape = NULL;
		uint2 * d_inbound_vertices = NULL, *d_outbound_vertices, *d_shape;

		uint32_t *d_outbound, *d_inbound, *d_weights;

		generator(aGraph, 100, 0, 100, 0.7, false);
		//cin >> aGraph;

		// write to stdout
		cout << aGraph << endl;

		cudaSetup(aGraph, inbound_vertices, outbound_vertices, shape);

		const uint32_t V = shape->x;
		const uint32_t E = shape->y;

		uint32_t *outbound = new uint32_t[V];
		uint32_t *inbound = new uint32_t[V];
		uint32_t *weights = new uint32_t[V];

		// start node
		uint32_t current_node = 0, *d_current_node = NULL;

		std::fill_n(weights, V, UINT32_MAX);

		uint32_t *d_num_edges, *d_idx_edges, *d_target, *d_weight;

		hipMalloc(&d_inbound_vertices, E * 2 * sizeof(uint2));
		hipMalloc(&d_outbound_vertices, V * sizeof(uint2));
		hipMalloc(&d_shape, sizeof(uint2));

		hipMalloc(&d_outbound, V * sizeof(uint32_t));
		hipMalloc(&d_inbound, V * sizeof(uint32_t));
		hipMalloc(&d_weights, V * sizeof(uint32_t));
		hipMalloc(&d_current_node, sizeof(uint32_t));

		hipMemcpy(d_inbound_vertices, inbound_vertices, E * 2 * sizeof(uint2), hipMemcpyHostToDevice);
		hipMemcpy(d_outbound_vertices, outbound_vertices, V * sizeof(uint2), hipMemcpyHostToDevice);
		hipMemcpy(d_shape, shape, sizeof(uint2), hipMemcpyHostToDevice);

		hipMemcpy(d_outbound, outbound, V * sizeof(uint32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_inbound, inbound, V * sizeof(uint32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_weights, weights, V * sizeof(uint32_t), hipMemcpyHostToDevice);
		hipMemcpy(d_current_node, &current_node, sizeof(uint32_t), hipMemcpyHostToDevice);

		begin = chrono::steady_clock::now();
		mst << <1, 1024 >> > (d_inbound_vertices, d_outbound_vertices, d_shape, d_inbound, d_outbound, d_weights, d_current_node);
		end = chrono::steady_clock::now();
		runtime = (chrono::duration_cast<chrono::duration<double>>(end - begin)).count() * 1000;

		cout << runtime << endl;

		hipMemcpy(outbound, d_outbound, V * sizeof(uint32_t), hipMemcpyDeviceToHost);
		hipMemcpy(inbound, d_inbound, V * sizeof(uint32_t), hipMemcpyDeviceToHost);
		hipMemcpy(weights, d_weights, V * sizeof(uint32_t), hipMemcpyDeviceToHost);

		print_result(outbound, inbound, weights, V);

		hipFree(d_inbound_vertices);
		hipFree(d_outbound_vertices);
		hipFree(d_shape);

		hipFree(d_inbound);
		hipFree(d_outbound);
		hipFree(d_weights);
		hipFree(d_current_node);

		delete[] inbound_vertices;
		delete[] outbound_vertices;
		delete[] shape;

		delete[] inbound;
		delete[] outbound;
		delete[] weights;
	}
}
