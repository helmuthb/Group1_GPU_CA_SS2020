#include "hip/hip_runtime.h"
//
// CUDA implemenation of Prim's Minimum Spanning Tree Algorithm
//
//
// Please refer to the report for documentation on all the data structures used
// here, as well as an outline of the implementation.
//


#include <cmath>

#include "cuda_prim.hpp"


//////////////////////////
// Options
//////////////////////////

#define BLOCKSIZE 1024


//
// Initialize the compact adjacency list representation (Wang et al.)
// 
void cudaSetup(const Graph& g, uint2 *vertices, uint2 *edges)
{
    uint32_t num_vertices = g.num_vertices();

    // Calculate data for each vertex, and the edges to its neighbors 
    for (uint32_t v = 0; v < num_vertices; ++v) {
        std::vector<EdgeTarget> neighbors;
        g.neighbors(v, neighbors);

        // Store vertex neighbor count and offset
        vertices[v].x = neighbors.size();
        vertices[v].y = 0;
        if (v == 0) {
            // Base case
            vertices[v].y = 0;
        } else {
            // Current offset = previous offset + number of previous nodes
            vertices[v].y = vertices[v-1].y + vertices[v-1].x;
        }

        // Store each edge, starting at the previously computed offset
        uint32_t idx = vertices[v].y;
        for (auto nb = neighbors.begin(); nb < neighbors.end(); ++nb) {
            // Store the neighbor vertex ID, and the weight of the edge
            edges[idx].x = nb->vertex_to;
            edges[idx].y = nb->weight;
            idx++;
        }
    }
}


//
// Kernel implementing the swap operation
//
//
__global__ void mst_swap_and_next(uint32_t *outbound, uint32_t *inbound, uint32_t *weights,
                                  uint32_t *tmp_best, uint32_t *current_vertex)
{
    uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t best = *tmp_best;
    if (idx == 0) {
        // No need to swap if the best edge is already at the front
        if (best != 0) {
            uint32_t outA = outbound[0];
            uint32_t inA  = inbound[0];
            uint32_t wA   = weights[0];
            uint32_t outB = outbound[best];
            uint32_t inB  = inbound[best];
            uint32_t wB   = weights[best];

            outbound[0]    = outB;
            inbound[0]     = inB;
            weights[0]     = wB;
            outbound[best] = outA;
            inbound[best]  = inA;
            weights[best]  = wA;
        }

        *current_vertex = inbound[0];
    }
}


//
// Kernel implementing the weight update primitive
//
// Uses the compact adjacency list as read-only input, and writes to the three
// MST data structures. Each thread accesses only one "row" of the MST data
// structure, so there is no need to synchronize anything.
//
// current_vertex points to the ID of the vertex from which the new paths are to be
// checked, and num_remaining is the offset of the not-yet-fixed edges.
//  
__global__ void mst_update(uint2 *vertices, uint2 *edges,
                           uint32_t *outbound, uint32_t *inbound, uint32_t *weights,
                           uint32_t *current_vertex, uint32_t num_remaining)
{
    uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    // TODO: vdata and edata are identical for all threads executing the
    // update,  these could be cached in shared mem!

    if (idx < num_remaining) {
        uint32_t other_vertex = inbound[idx];

        // Get edge offset and count for the current vertex
        // .x = count, .y = offset
        uint2 vdata = vertices[*current_vertex];

        // Iterate from offset to offset+count to find the weight from
        // current_vertex to other_vertex (if it exists)
        for (uint32_t i = vdata.y; i < vdata.y + vdata.x; ++i) {
            uint2 edata = edges[i];
            if (edata.x == other_vertex) {
                // If this edge provides a route to v better than the previously known one, replace it
                if (edata.y < weights[idx]) {
                    outbound[idx] = *current_vertex;
                    weights[idx] = edata.y;
                }
            }
        }
    }
}


//
// Kernel implementing the min reduction primitive
//
// indices:
//   Use NULL in the first step of the reduction    => SETS the index
//   Use non-NULL as input to the second reduction  => CARRIES over the index
//
__global__ void mst_minweight(uint32_t *indices, uint32_t *weights,
                              uint32_t *tmp_best, uint32_t *tmp_minweights,
                              uint32_t num_remaining)
{
    uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    // TODO: This is the vanilla, totally un-optimized version of the
    // reduction! Once this is working, adapt as per the NVIDIA slides

    // Store the per-thread best index and minimum weight
    __shared__ uint32_t shm_best[BLOCKSIZE];
    __shared__ uint32_t shm_minweights[BLOCKSIZE];

    if (idx < num_remaining) {
        // Each thread loads one element from global to shared memory (indices optional)
        if (indices == NULL) {
            shm_best[threadIdx.x] = idx;
        } else {
            shm_best[threadIdx.x] = indices[idx];
        }
        shm_minweights[threadIdx.x] = weights[idx]; 

        __syncthreads();

        // Perform the reduction, as per NVIDIA guidelines
        for (uint32_t s = 1; s < blockDim.x; s *= 2) {
            uint32_t left = 2 * s * threadIdx.x;

            if (left < blockDim.x) {
                uint32_t right = left + s;
                // Only compare if the counterpart is still within bounds
                if (right + (blockDim.x * blockIdx.x) < num_remaining) {
                    // If the best weight is not already at position ti, move it there
                    if (shm_minweights[right] < shm_minweights[left]) {
                        shm_best[left] = shm_best[right];
                        shm_minweights[left] = shm_minweights[right];
                    }
                }
            }
            __syncthreads();
        }

        // The last active thread of the block writes the result back
        if (threadIdx.x == 0) {
            tmp_best[blockIdx.x] = shm_best[0];
            tmp_minweights[blockIdx.x] = shm_minweights[0];
        }
    }
}


//
// Entry point for CUDA Prim's Algorithm
//
// This uses:
//   * Compact Adjacency List as proposed by Wang et al., based on Harish et al.
//   * MST data structure as proposed by Wang et al.
//
void cudaPrimAlgorithm(uint2 *vertices, uint32_t num_vertices,
                       uint2 *edges, uint32_t num_edges,
                       uint32_t *outbound, uint32_t *inbound, uint32_t *weights) {

    // Initialize the MST data structure
    for (uint32_t i = 0; i < num_vertices - 1; ++i) {
        outbound[i] = 0;
        inbound[i] = i + 1;
        weights[i] = Graph::WEIGHT_INFTY;
    }

    // Data structures in device memory
    uint2 *d_vertices, *d_edges;
    uint32_t *d_outbound, *d_inbound, *d_weights;
    // Temporary result storage
    uint32_t *d_tmp_best, *d_tmp_minweights, *d_current_vertex;

    if (BLOCKSIZE == 1) {
        throw new std::out_of_range("BLOCKSIZE must be greater than 1");
    }
    else if (ceil(log2(BLOCKSIZE)) != floor(log2(BLOCKSIZE))) {
        throw new std::out_of_range("BLOCKSIZE must be a power of 2");
    }

    // Total number of blocks needed to process all edges (one thread per edge)
    uint32_t total_blocks = static_cast<uint32_t>(std::ceil(static_cast<float>(num_vertices-1) / BLOCKSIZE));
    if (total_blocks > BLOCKSIZE) {
        throw new std::out_of_range("Cannot reduce more than BLOCKSIZE blocks");
    }

    // Allocate memory for the data structures in device memory
    hipMalloc(&d_vertices,       num_vertices     * sizeof(uint2));
    hipMalloc(&d_edges,          num_edges        * sizeof(uint2));
    hipMalloc(&d_outbound,       (num_vertices-1) * sizeof(uint32_t));
    hipMalloc(&d_inbound,        (num_vertices-1) * sizeof(uint32_t));
    hipMalloc(&d_weights,        (num_vertices-1) * sizeof(uint32_t));
    hipMalloc(&d_tmp_best,       total_blocks     * sizeof(uint32_t));
    hipMalloc(&d_tmp_minweights, total_blocks     * sizeof(uint32_t));
    hipMalloc(&d_current_vertex, 1                * sizeof(uint32_t));

    // Transfer inputs to device memory
    hipMemcpy(d_vertices, vertices, num_vertices     * sizeof(uint2),    hipMemcpyHostToDevice);
    hipMemcpy(d_edges,    edges,    num_edges        * sizeof(uint2),    hipMemcpyHostToDevice);
    hipMemcpy(d_outbound, outbound, (num_vertices-1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_inbound,  inbound,  (num_vertices-1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_weights,  weights,  (num_vertices-1) * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemset(d_tmp_best,       0,  total_blocks     * sizeof(uint32_t));
    hipMemset(d_tmp_minweights, 0,  total_blocks     * sizeof(uint32_t));
    hipMemset(d_current_vertex, 0,  1                * sizeof(uint32_t));

    for (uint32_t remaining_offset = 0; remaining_offset < num_vertices - 1; ++remaining_offset) {
        uint32_t num_remaining        = num_vertices - 1 - remaining_offset;
        uint32_t num_remaining_blocks = static_cast<uint32_t>(std::ceil(static_cast<float>(num_remaining) / BLOCKSIZE));

        mst_update <<<num_remaining_blocks, BLOCKSIZE>>> (
                d_vertices, d_edges,
                d_outbound+remaining_offset, d_inbound+remaining_offset, d_weights+remaining_offset,
                d_current_vertex, num_remaining);

        // Invoke 1: minimum per block, stored in temporary result
        mst_minweight <<<num_remaining_blocks, BLOCKSIZE>>> (
                // Let minweight index the data
                NULL, 
                // Each iteration, we move forward in the MST list
                d_weights+remaining_offset,
                // But not in the temporary results list!
                d_tmp_best, d_tmp_minweights,
                num_remaining);

        // Invoke 2:
        // If we have more than one block, find minimum of all blocks
        if (num_remaining_blocks > 1) {
            mst_minweight <<<1, num_remaining_blocks>>> (
                    d_tmp_best, d_tmp_minweights,
                    d_tmp_best, d_tmp_minweights,
                    num_remaining_blocks);
        }

        // If the best edge is not at the beginning, we must swap edges
        mst_swap_and_next <<<1, 1>>> (
                d_outbound+remaining_offset, d_inbound+remaining_offset, d_weights+remaining_offset,
                &d_tmp_best[0], d_current_vertex);
    } // Outer loop

    // Copy the results back to host memory
    hipMemcpy(outbound, d_outbound, (num_vertices-1) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(inbound,  d_inbound,  (num_vertices-1) * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(weights , d_weights,  (num_vertices-1) * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_vertices);
    hipFree(d_edges);
    hipFree(d_inbound);
    hipFree(d_outbound);
    hipFree(d_weights);
    hipFree(d_tmp_best);
    hipFree(d_tmp_minweights);
    hipFree(d_current_vertex);
}
