#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>

#include "list_graph.hpp"
#include "generator.hpp"
#include "graph.hpp"
#include "print_helper.cu"

#include <chrono>


#define NUM_RUNS 1
#define SHM_FACTOR 2

#define NUM_VERTICES 513
#define DENSITY 0.2
#define MIN_WEIGHT 0
#define MAX_WEIGHT 50

using namespace std;

__global__ void min_reduction1(uint32_t *inbound, uint32_t *weights, uint2 *v_red) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	extern __shared__ uint2 shm[];

	shm[threadIdx.x].x = idx;
	shm[threadIdx.x + blockDim.x].x = idx + blockDim.x;
	shm[threadIdx.x].y = idx < NUM_VERTICES && inbound[idx] > NUM_VERTICES ? weights[idx] : UINT32_MAX;
	shm[threadIdx.x + blockDim.x].y = UINT32_MAX;
	
	__syncthreads();

	for (int j = blockDim.x * SHM_FACTOR; j > 1; j /= 2) {
		for (int k = 0; k < SHM_FACTOR; k++) {
			if (shm[threadIdx.x].y > shm[threadIdx.x + j / 2].y) {
				shm[threadIdx.x].x = shm[threadIdx.x + j / 2].x;
				shm[threadIdx.x].y = shm[threadIdx.x + j / 2].y;
			}
		}
	}
	__syncthreads();

	if (threadIdx.x == 0) {
		v_red[blockIdx.x].x = shm[0].x;
		v_red[blockIdx.x].y = shm[0].y;
	}
}

__global__ void min_reduction2(uint2 *v_red, uint32_t *current_node, uint32_t *last_node, uint32_t red1_blocks) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (red1_blocks == 1) {
		if (idx == 1) {
			*last_node = *current_node;
			*current_node = v_red[0].x;
		}
		return;
	}

	uint32_t half_size = red1_blocks / 2;

	for (int j = half_size; j > 1; j /= 2) {
		for (int i = 0; i < j; i += blockDim.x) {
			if (idx + i < j) {
				if (v_red[idx + i + j].y < v_red[idx + i].y) {
					v_red[idx + i].x = v_red[idx + i + j].x;
					v_red[idx + i].y = v_red[idx + i + j].y;
				}
			}
		}
		__syncthreads();
	}
	if (idx == 0) {
		*last_node = *current_node;
		if (v_red[1].y < v_red[0].y) {
			*current_node = v_red[1].x;
		}
		else {
			*current_node = v_red[0].x;
		}
	}
}

__global__ void update_mst(uint2 *outbound_vertices, uint2 *inbound_vertices, uint32_t *outbound, uint32_t *weights, uint32_t *current_node) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	uint32_t start_index = outbound_vertices[*current_node].y;
	uint32_t end_index = start_index + outbound_vertices[*current_node].x;

	if (idx < end_index - start_index) {
		uint32_t edge_idx = idx + start_index;
		if (inbound_vertices[edge_idx].y < weights[inbound_vertices[edge_idx].x]) {
			weights[inbound_vertices[edge_idx].x] = inbound_vertices[edge_idx].y;
			outbound[inbound_vertices[edge_idx].x] = *current_node;
		}
	}
}

__global__ void update_mst2(uint32_t *outbound, uint32_t *inbound, uint32_t *weights, uint32_t *current_node, uint32_t *last_node) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx == 0) {
		outbound[*last_node] = outbound[*current_node];
		inbound[*last_node] = *current_node;
		weights[*last_node] = weights[*current_node];
		weights[*current_node] = UINT32_MAX;
	}
}


void cuda_setup(const Graph& g, uint2 *&inbound_vertices, uint2 *&outbound_vertices, uint2 *&shape) {
	shape = new uint2;
	shape->x = g.num_vertices();
	shape->y = g.num_edges();
	inbound_vertices = new uint2[shape->y * 2];
	outbound_vertices = new uint2[shape->x];
	uint32_t pos = 0;
	for (uint32_t v = 0; v < shape->x; ++v) {
		std::vector<EdgeTarget> neighbors;
		g.neighbors(v, neighbors);
		outbound_vertices[v].x = neighbors.size();
		outbound_vertices[v].y = v == 0 ? 0 : v == 1 ? outbound_vertices[v - 1].x : outbound_vertices[v - 1].y + outbound_vertices[v - 1].x;
		for (auto nb = neighbors.begin(); nb < neighbors.end(); ++nb) {
			inbound_vertices[pos].x = nb->vertex_to;
			inbound_vertices[pos++].y = nb->weight;
		}
	}
}

void allocate_resources(uint2 *& inbound_vertices, uint2 *& outbound_vertices, uint2 *&shape, uint2 *& d_inbound_vertices, uint2 *& d_outbound_vertices, uint2 *&d_shape, uint2 *&d_red_array, uint32_t *outbound, uint32_t *inbound, uint32_t *weights, uint32_t current_node, uint32_t *&d_outbound, uint32_t *&d_inbound, uint32_t *&d_weights, uint32_t *&d_current_node, uint32_t *&d_last_node, uint32_t num_blocks) {
	hipMalloc(&d_inbound_vertices, shape->y * 2 * sizeof(uint2));
	hipMalloc(&d_outbound_vertices, shape->x * sizeof(uint2));
	hipMalloc(&d_shape, sizeof(uint2));

	hipMalloc(&d_outbound, shape->x * sizeof(uint32_t));
	hipMalloc(&d_inbound, shape->x * sizeof(uint32_t));
	hipMalloc(&d_weights, shape->x * sizeof(uint32_t));
	hipMalloc(&d_current_node, sizeof(uint32_t));
	hipMalloc(&d_last_node, sizeof(uint32_t));

	hipMalloc(&d_red_array, num_blocks * sizeof(uint2));

	hipMemcpy(d_inbound_vertices, inbound_vertices, shape->y * 2 * sizeof(uint2), hipMemcpyHostToDevice);
	hipMemcpy(d_outbound_vertices, outbound_vertices, shape->x * sizeof(uint2), hipMemcpyHostToDevice);
	hipMemcpy(d_shape, shape, sizeof(uint2), hipMemcpyHostToDevice);

	hipMemcpy(d_outbound, outbound, shape->x * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_inbound, inbound, shape->x * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, shape->x * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_current_node, &current_node, sizeof(uint32_t), hipMemcpyHostToDevice);
}

void free_resources(uint2 *& inbound_vertices, uint2 *& outbound_vertices, uint2 *&shape, uint2 *& d_inbound_vertices, uint2 *& d_outbound_vertices, uint2 *&d_shape, uint2 *&d_red_array, uint32_t *outbound, uint32_t *inbound, uint32_t *weights, uint32_t *&d_outbound, uint32_t *&d_inbound, uint32_t *&d_weights, uint32_t *&d_current_node, uint32_t *&d_last_node) {
	hipFree(d_inbound_vertices);
	hipFree(d_outbound_vertices);
	hipFree(d_shape);

	hipFree(d_inbound);
	hipFree(d_outbound);
	hipFree(d_weights);
	hipFree(d_current_node);
	hipFree(d_last_node);

	hipFree(d_red_array);

	delete[] inbound_vertices;
	delete[] outbound_vertices;
	delete[] shape;
}

uint32_t calc_num_blocks(uint32_t num_vertices) {
	if (num_vertices < 512) {
		return 0;
	}
	uint32_t sqr = sqrt(num_vertices);
	uint32_t factor = 1;
	while (sqr != 0) {
		sqr = sqr >> 1;
		factor++;
	}
	return factor;
}

void print_result(uint32_t * outbound, uint32_t *inbound, uint32_t *weights, uint32_t V) {
	cout << "H " << V << " " << V - 1 << " " << 1 << endl;
	uint32_t counter = 0;
	for (int i = 0; i < NUM_VERTICES; i++) {
		if (inbound[i] < NUM_VERTICES) {
			cout << "E " << outbound[i] << " " << inbound[i] << " " << weights[i] << endl;
			counter++;
		}
	}
	cout << "NUMBER LINES : " << counter << endl;
}

void print_raw(uint32_t * outbound, uint32_t *inbound, uint32_t *weights) {
	cout << "H " << NUM_VERTICES << " " << NUM_VERTICES << " " << 1 << endl;
	uint32_t counter = 0;
	for (int i = 0; i < NUM_VERTICES; i++) {
			cout << "E " << outbound[i] << " " << inbound[i] << " " << weights[i] << endl;
	}
}

int main()
{
	ListGraph g;

	chrono::steady_clock::time_point begin, end;
	double runtime;

	uint2 * inbound_vertices, *outbound_vertices, *shape = NULL;
	uint2 * d_inbound_vertices = NULL, *d_outbound_vertices = NULL, *d_shape = NULL;

	uint32_t *d_outbound = NULL, *d_inbound = NULL, *d_weights = NULL;

	generator(g, NUM_VERTICES, MIN_WEIGHT, MAX_WEIGHT, DENSITY, false, UINT64_MAX);
	//cin >> g;

	// write to stdout
	//cout << g << endl;

	for (int i = 0; i < NUM_RUNS; i++) {

		cuda_setup(g, inbound_vertices, outbound_vertices, shape);

		uint32_t outbound[NUM_VERTICES];
		uint32_t inbound[NUM_VERTICES];
		uint32_t weights[NUM_VERTICES];
		uint2 *d_red_array = NULL;

		fill(weights, weights + NUM_VERTICES, UINT32_MAX);

		// start node
		uint32_t current_node = 0, *d_current_node = NULL, *d_last_node = NULL;

		/*
		cout << "outbound:" << endl;
		for (int i = 0; i < shape->x; i++) {
			printf("%d %d\n", outbound_vertices[i].y, outbound_vertices[i].x);
		}
		cout << "inbound:" << endl;
		for (int i = 0; i < shape->y * 2; i++) {
			printf("%d %d\n", inbound_vertices[i].y, inbound_vertices[i].x);
		}
		cout << "shape:" << endl;
		cout << "Number of Vertices: " << shape[0].x << endl << "Number of edges: " << shape[0].y << endl;
		*/

		uint32_t num_blocks_factor = calc_num_blocks(NUM_VERTICES);

		uint32_t num_blocks = 1 << num_blocks_factor;
		uint32_t num_threads = num_blocks_factor == 0 ? NUM_VERTICES : 1 << (num_blocks_factor - 2);

		allocate_resources(inbound_vertices, outbound_vertices, shape, d_inbound_vertices, d_outbound_vertices, d_shape, d_red_array, outbound, inbound, weights, current_node, d_outbound, d_inbound, d_weights, d_current_node, d_last_node, num_blocks);

		cout << "NUM BLOCKS " << num_blocks << "NUM THREADS " << num_threads << endl;
		uint32_t shm_size = num_threads * sizeof(uint2) * SHM_FACTOR;

		begin = chrono::steady_clock::now();
		for (int i = 0; i < NUM_VERTICES - 1; i++) {
			update_mst << <num_blocks, num_threads >> > (d_outbound_vertices, d_inbound_vertices, d_outbound, d_weights, d_current_node);
			min_reduction1 << <num_blocks, num_threads, shm_size >> > (d_inbound, d_weights, d_red_array);
			min_reduction2 << <1, num_threads >> > (d_red_array, d_current_node, d_last_node, num_blocks);
			update_mst2 << <num_blocks, num_threads >> > (d_outbound, d_inbound, d_weights, d_current_node, d_last_node);
		}
		end = chrono::steady_clock::now();
		runtime = (chrono::duration_cast<chrono::duration<double>>(end - begin)).count() * 1000;

		hipMemcpy(outbound, d_outbound, shape->x * sizeof(uint32_t), hipMemcpyDeviceToHost);
		hipMemcpy(inbound, d_inbound, shape->x * sizeof(uint32_t), hipMemcpyDeviceToHost);
		hipMemcpy(weights, d_weights, shape->x * sizeof(uint32_t), hipMemcpyDeviceToHost);

		print_result(outbound, inbound, weights, shape->x);
		//print_raw(outbound, inbound, weights);

		cout << runtime << " milliseconds." << endl;

		free_resources(inbound_vertices, outbound_vertices, shape, d_inbound_vertices, d_outbound_vertices, d_shape, d_red_array, outbound, inbound, weights, d_outbound, d_inbound, d_weights, d_current_node, d_last_node);

	}
}