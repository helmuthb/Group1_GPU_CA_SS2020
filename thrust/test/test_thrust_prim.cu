#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "doctest.h"
#include "thrust_prim.hpp"
#include "list_graph.hpp"
#include "generator.hpp"
#include <set>

TEST_CASE("Thrust prim for tiny graph") {
    ListGraph g(6, false);
    g.set(0,1,1);
    g.set(0,2,3);
    g.set(0,5,2);
    g.set(1,2,5);
    g.set(1,3,1);
    g.set(2,3,2);
    g.set(2,4,1);
    g.set(3,4,4);
    g.set(4,5,5);
    uint32_t V = g.num_vertices();
    uint32_t E = g.num_edges();
    thrust::host_vector<uint2> vertex_adjacent_count_index(V);
    thrust::host_vector<uint2> edge_target_weight(2*E);
    thrust::host_vector<uint32_t> mst_out(V);
    thrust::host_vector<uint32_t> mst_in(V);
    thrust::host_vector<uint32_t> mst_weight(V);
    thrustSetup(g, vertex_adjacent_count_index, edge_target_weight);
    thrustPrimAlgorithm(vertex_adjacent_count_index, edge_target_weight, mst_out, mst_in, mst_weight, V, E);
    // sum up weights
    int w = 0;
    for (int i=0; i<V-1; ++i) {
        w += mst_weight[i];
    }
    CHECK(w == 7);
    // check correct weight from original graph
    for (int i=0; i<V-1; ++i) {
        int f = mst_out[i];
        int t = mst_in[i];
        std::cout << f << "->" << t << " (" << mst_weight[i] << ")" << std::endl;
        CHECK(mst_weight[i] == g(f, t));
    }
    // check reachability of each node
    std::set<int> mst_nodes;
    mst_nodes.insert(mst_out[0]);
    for (int i=0; i<V-1; ++i) {
        // check if source node already in mst nodes
        CHECK(mst_nodes.count(mst_out[i]) == 1);
        // add new target node
        mst_nodes.insert(mst_in[i]);
    }
    CHECK(mst_nodes.size() == V);
}

TEST_CASE("Thrust prim for micro graph") {
    ListGraph g(3, false);
    g.set(0,1,1);
    g.set(0,2,2);
    g.set(1,2,3);
    uint32_t V = g.num_vertices();
    uint32_t E = g.num_edges();
    thrust::host_vector<uint2> vertex_adjacent_count_index(V);
    thrust::host_vector<uint2> edge_target_weight(2*E);
    thrust::host_vector<uint32_t> mst_out(V);
    thrust::host_vector<uint32_t> mst_in(V);
    thrust::host_vector<uint32_t> mst_weight(V);
    thrustSetup(g, vertex_adjacent_count_index, edge_target_weight);
    thrustPrimAlgorithm(vertex_adjacent_count_index, edge_target_weight, mst_out, mst_in, mst_weight, V, E);
    // sum up weights
    int w = 0;
    for (int i=0; i<V-1; ++i) {
        w += mst_weight[i];
    }
    CHECK(w == 3);
    // check correct weight from original graph
    for (int i=0; i<V-1; ++i) {
        int f = mst_out[i];
        int t = mst_in[i];
        CHECK(mst_weight[i] == g(f, t));
    }
    // check reachability of each node
    std::set<int> mst_nodes;
    mst_nodes.insert(mst_out[0]);
    for (int i=0; i<V-1; ++i) {
        // check if source node already in mst nodes
        CHECK(mst_nodes.count(mst_out[i]) == 1);
        // add new target node
        mst_nodes.insert(mst_in[i]);
    }
    CHECK(mst_nodes.size() == V);
}

TEST_CASE("Thrust prim for large graph") {
    ListGraph g(1000, false);
    generator(g, 1000, 2, 10, 0.5, false);
    int max_weight = 10 * 999;
    uint32_t V = g.num_vertices();
    uint32_t E = g.num_edges();
    thrust::host_vector<uint2> vertex_adjacent_count_index(V);
    thrust::host_vector<uint2> edge_target_weight(2*E);
    thrust::host_vector<uint32_t> mst_out(V);
    thrust::host_vector<uint32_t> mst_in(V);
    thrust::host_vector<uint32_t> mst_weight(V);
    thrustSetup(g, vertex_adjacent_count_index, edge_target_weight);
    thrustPrimAlgorithm(vertex_adjacent_count_index, edge_target_weight, mst_out, mst_in, mst_weight, V, E);
    // sum up weights
    int w = 0;
    for (int i=0; i<V-1; ++i) {
        w += mst_weight[i];
    }
    CHECK(w <= max_weight);
    // check correct weight from original graph
    int wrong_weights = 0;
    for (int i=0; i<V-1; ++i) {
        int f = mst_out[i];
        int t = mst_in[i];
        if (mst_weight[i] != g(f, t)) ++wrong_weights;
    }
    CHECK(wrong_weights == 0);
    // check reachability of each node
    std::set<int> mst_nodes;
    mst_nodes.insert(mst_out[0]);
    int nodes_missing = 0;
    for (int i=0; i<V-1; ++i) {
        // check if source node already in mst nodes
        if (mst_nodes.count(mst_out[i]) != 1) ++nodes_missing;
        // add new target node
        mst_nodes.insert(mst_in[i]);
    }
    CHECK(nodes_missing == 0);
    CHECK(mst_nodes.size() == V);
}
