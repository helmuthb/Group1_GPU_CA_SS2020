#include "hip/hip_runtime.h"
#include "graph.hpp"
#include "matrix_graph.hpp"
#include "sparse_graph.hpp"
#include "list_graph.hpp"
#include "thrust_prim.hpp"
#include "cuda_prim.hpp"
#include "generator.hpp"
#include "cpu_prim.hpp"
#include <chrono>
#include <iostream>
#ifdef WITH_BOOST
#include <boost/graph/prim_minimum_spanning_tree.hpp>
#endif

using namespace std::chrono;

double cudaRuntime(const Graph& g, int cntRuns) {
    steady_clock::time_point begin, end;
    double runtime;

    // prepare data for thrust
    uint2 * inbound_vertices, *outbound_vertices, *shape = NULL;
    cudaSetup(g, inbound_vertices, outbound_vertices, shape);
    const uint32_t V = shape->x;

    uint32_t *outbound = new uint32_t[V];
    uint32_t *inbound = new uint32_t[V];
    uint32_t *weights = new uint32_t[V];

    // allow for warm-up
    cudaPrimAlgorithm(inbound_vertices, outbound_vertices, shape,
        inbound, outbound, weights);

    // now the real test run
    begin = steady_clock::now();
    for (int i=0; i<cntRuns; ++i) {
        // find MST solution
        cudaPrimAlgorithm(inbound_vertices, outbound_vertices, shape,
            inbound, outbound, weights);
        }
    end = steady_clock::now();
    runtime = (duration_cast<duration<double>>(end-begin)).count();

    delete[] inbound_vertices;
    delete[] outbound_vertices;
    delete[] shape;

    delete[] inbound;
    delete[] outbound;
    delete[] weights;

    // return as miliseconds per round
    return 1000.*runtime/cntRuns;    
}

double thrustRuntime(const Graph& g, int cntRuns) {
    steady_clock::time_point begin, end;
    double runtime;

    // prepare data for thrust
    thrust::host_vector<uint32_t> num_edges;
    thrust::host_vector<uint32_t> idx_edges;
    thrust::host_vector<uint32_t> target;
    thrust::host_vector<int32_t> weight;
    thrustPrepare(g, &num_edges, &idx_edges, &target, &weight);
    thrust::host_vector<uint32_t> predecessor;
    // allow for warm-up
    thrustPrimAlgorithm(&num_edges, &idx_edges, &target, &weight, &predecessor);
    // now the real test run
    begin = steady_clock::now();
    for (int i=0; i<cntRuns; ++i) {
        // find MST solution
        thrustPrimAlgorithm(&num_edges, &idx_edges, &target, &weight, &predecessor);
    }
    end = steady_clock::now();
    runtime = (duration_cast<duration<double>>(end-begin)).count();
    // return as miliseconds per round
    return 1000.*runtime/cntRuns;    
}

template <class T_GRAPH>
double cpuRuntime(const Graph& g, int cntRuns) {
    steady_clock::time_point begin, end;
    double runtime;

    // allow for warm-up
    T_GRAPH mst;
    cpuPrimAlgorithm(g, mst);
    // now the real test run
    begin = steady_clock::now();
    for (int i=0; i<cntRuns; ++i) {
        MatrixGraph mst2;
        // find MST solution
        cpuPrimAlgorithm(g, mst2);
    }
    end = steady_clock::now();
    runtime = (duration_cast<duration<double>>(end-begin)).count();
    // return as miliseconds per round
    return 1000.*runtime/cntRuns;
}

#ifdef WITH_BOOST
struct do_nothing_dijkstra_visitor : boost::default_dijkstra_visitor {};

double boostRuntime(const Graph& g, int cntRuns) {
    steady_clock::time_point begin, end;
    BoostGraph boost_g;
    double runtime;

    // allow for warm-up
    g.toBoost(boost_g);
    auto p = std::vector<boost::graph_traits<BoostGraph>::vertex_descriptor >(g.num_vertices());
    boost::prim_minimum_spanning_tree(boost_g, &p[0]);
    begin = steady_clock::now();
    for (int i=0; i<cntRuns; ++i) {
        boost::prim_minimum_spanning_tree(boost_g, &p[0]);
    }
    end = steady_clock::now();
    runtime = (duration_cast<duration<double>>(end-begin)).count();
    // return as miliseconds per round
    return 1000.*runtime/cntRuns;
}
#endif

void runParamSet(std::ostream& os, int num_vertices, int weight_range, float density,
                 int numReplica, int cntRuns, uint64_t seed) {
    for (int i=0; i<numReplica; ++i) {
        // create an undirected graph, using a different seed in each replica
        ListGraph g;
        generator(g, num_vertices, 0, weight_range, density, false, seed+numReplica);
        // run through all implementations and get runtime
        double runtime;

        runtime = cpuRuntime<ListGraph>(g, cntRuns);
        // output to file 
        os << "cpu_l," << i
                << "," << num_vertices
                << "," << density
                << "," << weight_range
                << "," << runtime
                << std::endl;
/*
#ifdef WITH_BOOST
        // run through boost implementation
        runtime = boostRuntime(g, cntRuns);
        // output to file 
        os << "cpu_b," << i
                << "," << num_vertices
                << "," << density
                << "," << weight_range
                << "," << runtime
                << std::endl;
#endif
        // run through thrust implementation
        runtime = thrustRuntime(g, cntRuns);
        // output to file 
        os << "thrus," << i
                << "," << num_vertices
                << "," << density
                << "," << weight_range
                << "," << runtime
                << std::endl;
*/
        // run through cuda implementation
        runtime = cudaRuntime(g, cntRuns);
        // output to file 
        os << "cuda," << i
                << "," << num_vertices
                << "," << density
                << "," << weight_range
                << "," << runtime
                << std::endl;
    }
}

int main(int argc, char* argv[]) {
    std::cout << "implementation,vertices,density,weight_range,runtime" << std::endl;
    runParamSet(std::cout, 10000, 50,  0.01, 3, 1, 42);
    runParamSet(std::cout, 50000, 50, 0.001, 3, 1, 42);
}
